#include "hip/hip_runtime.h"
#include "CudaMatrixOperations.h"
#include "hip/hip_runtime.h"
#include ""

const int kThreadsPerBlockDimension = 32;

//So as far as I can tell, CUDAs memcpy doesnt support deep copying, so as a work around, I copy all of the data (at this point only row and column count) into the device memory and then create a second variable for the matrix data itself
//Input is the Cuda matrix from the cpu, device matrix data is an out parameter that is the pointer to the memory on the device. Sorry if this comment isn't helpful

CudaMatrix* putMatrixIntoDeviceMemory(const CudaMatrix& input, double** deviceMatrixData)
{
	CudaMatrix* result; 
	hipMalloc((void**)&result , sizeof(CudaMatrix));
	hipMemcpy((void*)result, (void*)&input, sizeof(CudaMatrix), hipMemcpyHostToDevice);

	hipMalloc((void**)deviceMatrixData, input.rowCount * input.columnCount * sizeof(double));
	hipMemcpy(*deviceMatrixData, input.data, input.rowCount * input.columnCount * sizeof(double), hipMemcpyHostToDevice);
	return result;
}

__global__ void matrixMultiplicationKernel(double* result, const double* firstMatrixData, const double* secondMatrixData, const CudaMatrix* firstMatrixDimensions, const CudaMatrix* secondMatrixDimensions)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (row < firstMatrixDimensions->rowCount && column < secondMatrixDimensions->columnCount)
	{
		for (int i = 0; i < firstMatrixDimensions->columnCount; i++)
		{
			result[(row * secondMatrixDimensions->columnCount) + column] += firstMatrixData[(row * firstMatrixDimensions->columnCount) + i] * secondMatrixData[(i * secondMatrixDimensions->columnCount) + column];
		}
	}
}

__declspec(dllexport) CudaMatrix multiplyMatricesWithCuda(const CudaMatrix& firstMatrix, const CudaMatrix& secondMatrix)
{
	CudaMatrix result(firstMatrix.rowCount, secondMatrix.columnCount);
	
	double* deviceFirstMatrixData; 
	CudaMatrix* deviceFirstMatrixDimensions = putMatrixIntoDeviceMemory(firstMatrix, &deviceFirstMatrixData);

	double* deviceSecondMatrixData;
	CudaMatrix* deviceSecondMatrixDimensions = putMatrixIntoDeviceMemory(secondMatrix, &deviceSecondMatrixData);

	double* deviceResultMatrixData;
	hipMalloc((void**) &deviceResultMatrixData, result.rowCount * result.columnCount * sizeof(double));
	hipMemset(deviceResultMatrixData, 0.0, result.rowCount * result.columnCount * sizeof(double));

	dim3 blockCounts((result.columnCount + (kThreadsPerBlockDimension - 1)) / kThreadsPerBlockDimension, ((result.rowCount + (kThreadsPerBlockDimension - 1)) / kThreadsPerBlockDimension));
	matrixMultiplicationKernel<<<blockCounts, dim3(kThreadsPerBlockDimension, kThreadsPerBlockDimension)>>>(deviceResultMatrixData, deviceFirstMatrixData, deviceSecondMatrixData, deviceFirstMatrixDimensions, deviceSecondMatrixDimensions);

	hipMemcpy(result.data, deviceResultMatrixData, result.rowCount * result.columnCount * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(deviceFirstMatrixData);
	hipFree(deviceSecondMatrixData);
	hipFree(deviceResultMatrixData);
	hipFree(deviceFirstMatrixDimensions);
	hipFree(deviceSecondMatrixDimensions);

	return result;
}

__global__ void matrixAdditionKernel(double* result, const double* firstMatrixData, const double* secondMatrixData, const CudaMatrix* firstMatrixDimensions, const CudaMatrix* secondMatrixDimensions)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (row < firstMatrixDimensions->rowCount && column < firstMatrixDimensions->columnCount)
	{
		result[(row * firstMatrixDimensions->columnCount) + column] = firstMatrixData[(row * firstMatrixDimensions->columnCount) + column] + secondMatrixData[(row * secondMatrixDimensions->columnCount) + column];
	}
}

__declspec(dllexport) CudaMatrix addMatricesWithCuda(const CudaMatrix& firstMatrix, const CudaMatrix& secondMatrix)
{
	CudaMatrix result(firstMatrix.rowCount, firstMatrix.columnCount);

	double* deviceFirstMatrixData;
	CudaMatrix* deviceFirstMatrixDimensions = putMatrixIntoDeviceMemory(firstMatrix, &deviceFirstMatrixData);

	double* deviceSecondMatrixData;
	CudaMatrix* deviceSecondMatrixDimensions = putMatrixIntoDeviceMemory(secondMatrix, &deviceSecondMatrixData);

	double* deviceResultMatrixData;
	hipMalloc((void**)&deviceResultMatrixData, result.rowCount * result.columnCount * sizeof(double));
	hipMemset(deviceResultMatrixData, 0.0, result.rowCount * result.columnCount * sizeof(double));

	dim3 blockCounts((result.columnCount + (kThreadsPerBlockDimension - 1)) / kThreadsPerBlockDimension, ((result.rowCount + (kThreadsPerBlockDimension - 1)) / kThreadsPerBlockDimension));
	matrixAdditionKernel<<<blockCounts, dim3(kThreadsPerBlockDimension, kThreadsPerBlockDimension)>>> (deviceResultMatrixData, deviceFirstMatrixData, deviceSecondMatrixData, deviceFirstMatrixDimensions, deviceSecondMatrixDimensions);

	hipMemcpy(result.data, deviceResultMatrixData, result.rowCount * result.columnCount * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(deviceFirstMatrixData);
	hipFree(deviceSecondMatrixData);
	hipFree(deviceResultMatrixData);
	hipFree(deviceFirstMatrixDimensions);
	hipFree(deviceSecondMatrixDimensions);

	return result;
}

__global__ void matrixSubtractionKernel(double* result, const double* firstMatrixData, const double* secondMatrixData, const CudaMatrix* firstMatrixDimensions, const CudaMatrix* secondMatrixDimensions)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (row < firstMatrixDimensions->rowCount && column < firstMatrixDimensions->columnCount)
	{
		result[(row * firstMatrixDimensions->columnCount) + column] = firstMatrixData[(row * firstMatrixDimensions->columnCount) + column] - secondMatrixData[(row * secondMatrixDimensions->columnCount) + column];
	}
}

__declspec(dllexport) CudaMatrix subtractMatricesWithCuda(const CudaMatrix& firstMatrix, const CudaMatrix& secondMatrix)
{
	CudaMatrix result(firstMatrix.rowCount, firstMatrix.columnCount);

	double* deviceFirstMatrixData;
	CudaMatrix* deviceFirstMatrixDimensions = putMatrixIntoDeviceMemory(firstMatrix, &deviceFirstMatrixData);

	double* deviceSecondMatrixData;
	CudaMatrix* deviceSecondMatrixDimensions = putMatrixIntoDeviceMemory(secondMatrix, &deviceSecondMatrixData);

	double* deviceResultMatrixData;
	hipMalloc((void**)&deviceResultMatrixData, result.rowCount * result.columnCount * sizeof(double));
	hipMemset(deviceResultMatrixData, 0.0, result.rowCount * result.columnCount * sizeof(double));

	dim3 blockCounts((result.columnCount + (kThreadsPerBlockDimension - 1)) / kThreadsPerBlockDimension, ((result.rowCount + (kThreadsPerBlockDimension - 1)) / kThreadsPerBlockDimension));
	matrixSubtractionKernel<<<blockCounts, dim3(kThreadsPerBlockDimension, kThreadsPerBlockDimension)>>> (deviceResultMatrixData, deviceFirstMatrixData, deviceSecondMatrixData, deviceFirstMatrixDimensions, deviceSecondMatrixDimensions);

	hipMemcpy(result.data, deviceResultMatrixData, result.rowCount * result.columnCount * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(deviceFirstMatrixData);
	hipFree(deviceSecondMatrixData);
	hipFree(deviceResultMatrixData);
	hipFree(deviceFirstMatrixDimensions);
	hipFree(deviceSecondMatrixDimensions);

	return result;
}